#include "hip/hip_runtime.h"

#include "cudaCircles.cuh"
#include "../classes/shapes/circle.h"
//#include "opencv2/highgui.hpp"

//#include "opencv2/core/mat.hpp"

#define gpu_check(e)    \
if (e != hipSuccess) { \
    printf("cuda error - %d on %s:%d\n", e, __FILE__, __LINE__); \
    }

namespace vidga {
    namespace cuda {
        __device__ constexpr unsigned colRow2idx(unsigned col, unsigned row, unsigned sideLength) {
            return row * sideLength + col;
        }

        __device__ __always_inline void blendColors(float3 *pixel, float4 color, float modifier) {
            float circleModifier = color.w * modifier;
            float canvasModifier = 1 - circleModifier;
            pixel->x = min(color.x * circleModifier + pixel->x * canvasModifier, 1.f);
            pixel->y = min(color.y * circleModifier + pixel->y * canvasModifier, 1.f);
            pixel->z = min(color.z * circleModifier + pixel->z * canvasModifier, 1.f);
        }

        __global__  void genSmoothCircleMap(float *buffer, unsigned radius) {
            const unsigned int strideX = blockDim.x * gridDim.x;
            const unsigned int strideY = blockDim.y * gridDim.y;
            const unsigned int initialX = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int initialY = blockIdx.y * blockDim.y + threadIdx.y;
            int x = radius, y = radius;
            unsigned sideLength = radius * 2 + 1;
            for (unsigned row = initialX; row < sideLength; row += strideX) {
                int xValSq = (row - x) * (row - x);
                for (unsigned col = initialY; col < sideLength; col += strideY) {
                    int yValSq = (col - y) * (col - y);
                    float distance = sqrtf(xValSq + yValSq);
                    unsigned idx = colRow2idx(col, row, sideLength);
                    float diff = distance - radius;
                    if (diff < 0) {
                        buffer[idx] = 1.f;
                    } else if (diff < 1) {
                        buffer[idx] = 1.f - diff;
                    }
                }
            }
        }

        __global__ void
        drawUsingMap(float3 *buffer, unsigned width, unsigned height, const float *map, circle c) {
            const unsigned int strideX = blockDim.x * gridDim.x;
            const unsigned int strideY = blockDim.y * gridDim.y;
            const unsigned int posX = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int posY = blockIdx.y * blockDim.y + threadIdx.y;


            __shared__ unsigned mapShiftX, mapShiftY, left, top;
            if (c.center.x < c.radius) {
                left = 0;
                mapShiftX = c.radius - c.center.x;
            } else {
                left = c.center.x - c.radius;
                mapShiftX = 0;
            }

            if (c.center.y < c.radius) {
                top = 0;
                mapShiftY = c.radius - c.center.y;
            } else {
                top = c.center.y - c.radius;
                mapShiftY = 0;
            }

            unsigned right = min(width - 1, c.center.x + c.radius);
            unsigned bottom = min(height - 1, c.center.y + c.radius);

            for (unsigned col = top + posY; col <= bottom; col += strideY) {
                for (unsigned row = left + posX; row <= right; row += strideX) {
                    unsigned bufferIdx = colRow2idx(col, row, width);
                    float3 *pixel = &buffer[bufferIdx];

                    unsigned mapIdx = colRow2idx(col - top + mapShiftY, row - left + mapShiftX, c.radius * 2 + 1);
                    float modifier = map[mapIdx];
                    blendColors(pixel, c.color, modifier);
                }
            }
        }

/*
        __global__ void
        drawManyUsingMap(float3 *buffer, unsigned width, unsigned height, float **maps, unsigned mapsOffset,
                         circle *circles, unsigned nCircles) {
            const unsigned int strideX = blockDim.x * gridDim.x;
            const unsigned int strideY = blockDim.y * gridDim.y;
            const unsigned int posX = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int posY = blockIdx.y * blockDim.y + threadIdx.y;
            const unsigned int posZ = blockIdx.z * blockDim.z + threadIdx.z;

            unsigned mapShiftX, mapShiftY, left, top;
            circle *c;
            float *map;
            for (unsigned i = posZ; i < nCircles; i++) {
                c = &circles[i];
                map = maps[c->radius - mapsOffset];

                if (c->center.x < c->radius) {
                    left = 0;
                    mapShiftX = c->radius - c->center.x;
                } else {
                    left = c->center.x - c->radius;
                    mapShiftX = 0;
                }

                if (c->center.y < c->radius) {
                    top = 0;
                    mapShiftY = c->radius - c->center.y;
                } else {
                    top = c->center.y - c->radius;
                    mapShiftY = 0;
                }

                unsigned right = min(width - 1, c->center.x + c->radius);
                unsigned bottom = min(height - 1, c->center.y + c->radius);

                for (unsigned col = top + posY; col <= bottom; col += strideY) {
                    for (unsigned row = left + posX; row <= right; row += strideX) {
                        unsigned bufferIdx = colRow2idx(col, row, width);
                        float3 *pixel = &buffer[bufferIdx];

                        unsigned mapIdx = colRow2idx(col - top + mapShiftY, row - left + mapShiftX, c->radius * 2 + 1);
                        float modifier = map[mapIdx];
                        blendColors(pixel, c->color, modifier);
                    }
                }
            }
        }
*/

        void
        drawUsingMapHostFn(float3 *buffer, unsigned width, unsigned height, const float *map, circle c) {
            dim3 threads(16, 16, 1);
            dim3 blocks(4, 4, 1);
            drawUsingMap<<<blocks, threads>>>(buffer, width, height, map, std::move(c));
        }

 /*       void
        drawManyUsingMapHostFn(float3 *buffer, unsigned width, unsigned height, float **maps, unsigned mapsOffset,
                               const circle *circles, unsigned nCircles) {
            dim3 threads(16, 16, 1);
            dim3 blocks(1, 1, 1);
            circle* circlesGpu;
            size_t byteSize = sizeof(circle) * nCircles;
            auto e = hipMalloc(&circlesGpu, byteSize);
            auto e2 = hipMemcpy((void *) circlesGpu, circles, byteSize, hipMemcpyHostToDevice);
            printf("e %d, e2 %d\n", e, e2);
            drawManyUsingMap<<<blocks, threads>>>(buffer, width, height, maps, mapsOffset, circlesGpu, nCircles);
            gpu_check(hipGetLastError());
        }*/

        void initCircleMaps(unsigned minRadius, unsigned maxRadius, float ***gpuBuffers) {
            unsigned numCircles = maxRadius - minRadius + 1;
            *gpuBuffers = static_cast<float **>(malloc(numCircles * sizeof(float *)));
            for (auto i = minRadius; i <= maxRadius; i++) {
                auto idx = i - minRadius;
                auto winSideLength = 2 * i + 1;
                auto winPixels = winSideLength * winSideLength;
                unsigned memToAlloc = winPixels * sizeof(float);
                float **circleBuf = &(*gpuBuffers)[idx];
                hipMalloc(circleBuf, memToAlloc);
                genSmoothCircleMap<<<32, 32>>>(*circleBuf, i);
            }
            gpu_check(hipDeviceSynchronize());
        }

        void setGpuMatTo(float3 *mat, unsigned width, unsigned height, float val) {
            auto size = width * height * sizeof(float) * 3;
            hipMemset(mat, val, size);
        }

        float3 *getWhiteGpuMat(unsigned width, unsigned height) {
            auto size = width * height * sizeof(float) * 3;
            float3 *ret;
            gpu_check(hipMalloc(&ret, size));
            setGpuMatTo(ret, width, height, 1.f);
            return ret;
        }
    }
}