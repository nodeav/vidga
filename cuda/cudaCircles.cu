#include "hip/hip_runtime.h"
#include "cudaCircles.cuh"
#include "../classes/shapes/circle.h"
//#include "opencv2/highgui.hpp"

//#include "opencv2/core/mat.hpp"
namespace vidga {
    namespace cuda {
        constexpr unsigned colRow2idx(unsigned col, unsigned row, unsigned sideLength) {
            return row * sideLength + col;
        }

        __device__ __always_inline void blendColors(float3 *pixel, float4 color, float modifier) {
            float finalModifier = color.w * modifier;
            pixel->x = max(color.x * finalModifier + pixel->x, 1.f);
            pixel->y = max(color.y * finalModifier + pixel->y, 1.f);
            pixel->z = max(color.z * finalModifier + pixel->z, 1.f);
        }

        __global__ void genSmoothCircleMap(float *buffer, unsigned short radius) {
            const unsigned int strideX = blockDim.x * gridDim.x;
            const unsigned int strideY = blockDim.y * gridDim.y;
            const unsigned int initialX = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int initialY = blockIdx.y * blockDim.y + threadIdx.y;
            unsigned short x = radius, y = radius;
            unsigned short sideLength = radius * 2;
            for (unsigned row = initialX; row < sideLength; row += strideX) {
                int yValueSquared = (row - y) * (row - y);
                for (unsigned col = initialY; col < sideLength; col += strideY) {
                    float distance = sqrtf(yValueSquared + (col - x) * (col - x));
                    unsigned idx = colRow2idx(col, row, sideLength);
                    float diff = distance - radius + 1;
                    if (diff < 0) {
                        buffer[idx] = 1;
                    } else if (diff < 1) {
                        buffer[idx] = 1 - diff;
                    }
                }
            }
        }

        __global__ void
        drawUsingMap(float3 *buffer, unsigned width, unsigned height, const float *map, circle c) {
            const unsigned int strideX = blockDim.x * gridDim.x;
            const unsigned int strideY = blockDim.y * gridDim.y;
            const unsigned int initialX = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int initialY = blockIdx.y * blockDim.y + threadIdx.y;

            unsigned short sideLength = c.radius * 2;
            unsigned maxX = min(width, c.center.x + c.radius);
            unsigned maxY = min(height, c.center.y + c.radius);
            for (unsigned row = initialX; row < maxX; row += strideX) {
                for (unsigned col = initialY; col < maxY; col += strideY) {
                    unsigned idx = colRow2idx(col, row, sideLength);
                    float3 *pixel = &buffer[idx];
                    float modifier = map[idx];
                    blendColors(pixel, c.color, modifier);
                }
            }
        }

        void
        drawUsingMapHostFn(float3 *buffer, unsigned width, unsigned height, const float *map, circle c) {
            dim3 threads(8, 8, 1);
            dim3 blocks(1, 1, 1);
            drawUsingMap<<<blocks, threads>>>(buffer, width, height, map, c);
        }

        void initCircleMaps(unsigned minRadius, unsigned maxRadius, float **gpuBuffers) {
            unsigned numCircles = maxRadius - minRadius + 1;
            gpuBuffers = static_cast<float **>(malloc(numCircles * sizeof(float *)));
            for (auto i = minRadius; i <= maxRadius; i++) {
                auto idx = i - minRadius;
                unsigned memToAlloc = 4 * i * i * sizeof(float *);
                float *circleBuf = gpuBuffers[idx];
                hipMalloc(&circleBuf, memToAlloc);
                genSmoothCircleMap<<<32, 32>>>(circleBuf, i);
            }
            hipDeviceSynchronize();

        }
    }
}